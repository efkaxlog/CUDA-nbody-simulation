#include "hip/hip_runtime.h"
#include "nbody.h"
#include <iostream>

float *deviceXpos, *deviceYpos, *deviceZpos;
float *deviceXforces, *deviceYforces, *deviceZforces;
float *deviceMasses;
int particlesMemorySize;
Simulation *sim;

void cudaCalculate() {
    calculateForcesCuda<<<32711, 1024>>>(
        deviceXpos, deviceYpos, deviceZpos,
        deviceXforces, deviceYforces, deviceZforces,
        deviceMasses, sim->particlesNumber);
}

__global__ void calculateForcesCuda(float *deviceXpos, float *deviceYpos, float *deviceZpos, 
                                   float *deviceXforces, float *deviceYforces, float *deviceZforces,
                                   float *deviceMasses, int particlesNumber) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < particlesNumber*particlesNumber) {
        int particleIndex = tid / particlesNumber;
        int otherParticleIndex = tid % particlesNumber;
        if (particleIndex != otherParticleIndex) {
            float particleXpos = deviceXpos[otherParticleIndex];
            float particleYpos = deviceYpos[otherParticleIndex];
            float particleZpos = deviceZpos[otherParticleIndex];
            float otherXpos = deviceXpos[particleIndex];
            float otherYpos = deviceYpos[particleIndex];
            float otherZpos = deviceZpos[particleIndex];
            float mass = deviceMasses[particleIndex];
            float distx = otherXpos - particleXpos; 
            float disty = otherYpos - particleYpos; 
            float distz = otherZpos - particleZpos;
            float distance = sqrt(distx*distx + disty*disty + distz*distz);
            float xForce = 10.0f * mass / distance * (otherXpos - particleXpos) * 0.01f; 
            float yForce = 10.0f * mass / distance * (otherYpos - particleYpos) * 0.01f; 
            float zForce = 10.0f * mass / distance * (otherZpos - particleZpos) * 0.01f; 
            deviceXforces[otherParticleIndex] += xForce;
            deviceYforces[otherParticleIndex] += yForce;
            deviceZforces[otherParticleIndex] += zForce;
        }
    }
}

void setupCuda(Simulation *simulation) {
    sim = simulation;
    particlesMemorySize = sim->particlesNumber * sizeof(float);
    prepareCuda();
}

void prepareCuda() {
    hipMalloc((void**)&deviceXpos, particlesMemorySize);
    hipMemcpy(deviceXpos, sim->xPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);

    hipMalloc((void**)&deviceYpos, particlesMemorySize);
    hipMemcpy(deviceYpos, sim->yPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceZpos, particlesMemorySize);
    hipMemcpy(deviceZpos, sim->yPositions.data(),
    particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceXforces, particlesMemorySize);
    hipMemcpy(deviceXforces, sim->xForces.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceYforces, particlesMemorySize);
    hipMemcpy(deviceYforces, sim->yForces.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceZforces, particlesMemorySize);
    hipMemcpy(deviceZforces, sim->zForces.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceMasses, particlesMemorySize);
    hipMemcpy(deviceMasses, sim->masses.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
}

void updateData() {
    hipMemcpy(sim->xForces.data(), deviceXforces,
        particlesMemorySize, hipMemcpyDeviceToHost);
    hipMemcpy(sim->yForces.data(), deviceYforces,
        particlesMemorySize, hipMemcpyDeviceToHost);
    hipMemcpy(sim->zForces.data(), deviceZforces,
        particlesMemorySize, hipMemcpyDeviceToHost);
    
    hipMemcpy(deviceXpos, sim->xPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMemcpy(deviceYpos, sim->yPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMemcpy(deviceZpos, sim->zPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
}

void cleanupCuda() {
     hipFree(deviceXpos);
     hipFree(deviceYpos);
     hipFree(deviceZpos);
     hipFree(deviceXforces);
     hipFree(deviceYforces);
     hipFree(deviceZforces);
     hipFree(deviceMasses);
}
