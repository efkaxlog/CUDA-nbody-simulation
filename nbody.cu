#include "hip/hip_runtime.h"
#include "nbody.h"
#include <iostream>
#include <math.h>

namespace cuda {
    float *deviceXpos, *deviceYpos, *deviceZpos;
    float *deviceXforces, *deviceYforces, *deviceZforces;
    float *deviceMasses;
    int particlesMemorySize;
    int blocks;
    const int threads = 64;
    int particlesNumber;
    Simulation *sim;
}

using namespace cuda;

void cudaCalculate() {
    calculateForcesCuda<<<blocks, threads>>>(
        deviceXpos, deviceYpos, deviceZpos,
        deviceXforces, deviceYforces, deviceZforces,
        particlesNumber);
}

__global__ void calculateForcesCuda(float *deviceXpos, float *deviceYpos, float *deviceZpos, 
                                    float *deviceXforces, float *deviceYforces, float *deviceZforces,
                                    int particlesNumber) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ float localPosX[threads];
    __shared__ float localPosY[threads];
    __shared__ float localPosZ[threads];
    float3 body = {deviceXpos[tid],
                   deviceYpos[tid],
                   deviceZpos[tid]};
    float3 force = {0.0f, 0.0f, 0.0f};
    const float mass = 0.0001f;
    const float G = 2.0f;

    for (unsigned int i=0; i<particlesNumber; i+=threads) {
        localPosX[threadIdx.x] = deviceXpos[i + threadIdx.x]; 
        localPosY[threadIdx.x] = deviceYpos[i + threadIdx.x]; 
        localPosZ[threadIdx.x] = deviceZpos[i + threadIdx.x]; 
        __syncthreads();

        for (unsigned int index=0; index<threads; index++) {
            float3 other = {localPosX[index], 
                            localPosY[index],
                            localPosZ[index]};
            float3 r = {body.x - other.x,
                        body.y - other.y,
                        body.z - other.z};
            float distance = sqrtf((r.x*r.x + r.y*r.y + r.z*r.z) + 0.01f);
            force.x += G * mass / distance * (other.x - body.x); 
            force.y += G * mass / distance * (other.y - body.y); 
            force.z += G * mass / distance * (other.z - body.z); 
        }
        __syncthreads();
    }
    deviceXforces[tid] += force.x; 
    deviceYforces[tid] += force.y;           
    deviceZforces[tid] += force.z;
}


void setupCuda(Simulation *simulation) {
    sim = simulation;
    particlesNumber = sim->particlesNumber;
    particlesMemorySize = particlesNumber * sizeof(float);
    blocks = particlesNumber / threads;
    prepareCuda();
}

void prepareCuda() {
    hipMalloc((void**)&deviceXpos, particlesMemorySize);
    hipMemcpy(deviceXpos, sim->xPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);

    hipMalloc((void**)&deviceYpos, particlesMemorySize);
    hipMemcpy(deviceYpos, sim->yPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceZpos, particlesMemorySize);
    hipMemcpy(deviceZpos, sim->zPositions.data(),
    particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceXforces, particlesMemorySize);
    hipMemcpy(deviceXforces, sim->xForces.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceYforces, particlesMemorySize);
    hipMemcpy(deviceYforces, sim->yForces.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceZforces, particlesMemorySize);
    hipMemcpy(deviceZforces, sim->zForces.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
}

void updateData() {
    hipMemcpy(sim->xForces.data(), deviceXforces,
        particlesMemorySize, hipMemcpyDeviceToHost);
    hipMemcpy(sim->yForces.data(), deviceYforces,
        particlesMemorySize, hipMemcpyDeviceToHost);
    hipMemcpy(sim->zForces.data(), deviceZforces,
        particlesMemorySize, hipMemcpyDeviceToHost);
    
    hipMemcpy(deviceXpos, sim->xPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    hipMemcpy(deviceYpos, sim->yPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    hipMemcpy(deviceZpos, sim->zPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
}

void cleanupCuda() {
     hipFree(deviceXpos);
     hipFree(deviceYpos);
     hipFree(deviceZpos);
     hipFree(deviceXforces);
     hipFree(deviceYforces);
     hipFree(deviceZforces);
     hipFree(deviceMasses);
}
