#include "hip/hip_runtime.h"
#include "nbody.h"
#include <iostream>
#include <math.h>

namespace cuda {
    float *deviceXpos, *deviceYpos, *deviceZpos;
    float *deviceXforces, *deviceYforces, *deviceZforces;
    float *deviceMasses;
    int particlesMemorySize;
    int blocks, threads;
    int particlesNumber;
    Simulation *sim;
}

using namespace cuda;

void cudaCalculate() {
    calculateForcesCuda<<<blocks, threads>>>(
        deviceXpos, deviceYpos, deviceZpos,
        deviceXforces, deviceYforces, deviceZforces,
        deviceMasses, particlesNumber);
}

__global__ void calculateForcesCuda(float *deviceXpos, float *deviceYpos, float *deviceZpos, 
                                   float *deviceXforces, float *deviceYforces, float *deviceZforces,
                                   float *deviceMasses, int particlesNumber) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float particleXpos = deviceXpos[tid];
    float particleYpos = deviceYpos[tid];
    float particleZpos = deviceZpos[tid];
    if (tid <= particlesNumber) {
        for (int index=0; index<particlesNumber; index++) {
            if (tid != index) {
                float otherXpos = deviceXpos[index];
                float otherYpos = deviceYpos[index];
                float otherZpos = deviceZpos[index];
                float mass = deviceMasses[index];
                float distx = particleXpos - otherXpos; 
                float disty = particleYpos - otherYpos;
                float distz = particleZpos - otherZpos; 
                float distance = sqrt((distx*distx + disty*disty + distz*distz) + 0.01f);
                deviceXforces[tid] += 10.0f * mass / distance * (otherXpos - particleXpos); 
                deviceYforces[tid] += 10.0f * mass / distance * (otherYpos - particleYpos); 
                deviceZforces[tid] += 10.0f * mass / distance * (otherZpos - particleZpos); 
            }
        }
    }
}

void setupCuda(Simulation *simulation, int threads) {
    sim = simulation;
    particlesNumber = sim->particlesNumber;
    particlesMemorySize = particlesNumber * sizeof(float);
    cuda::threads = threads;
    blocks = particlesNumber / threads + 1;
    prepareCuda();
}

void prepareCuda() {
    hipMalloc((void**)&deviceXpos, particlesMemorySize);
    hipMemcpy(deviceXpos, sim->xPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);

    hipMalloc((void**)&deviceYpos, particlesMemorySize);
    hipMemcpy(deviceYpos, sim->yPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceZpos, particlesMemorySize);
    hipMemcpy(deviceZpos, sim->zPositions.data(),
    particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceXforces, particlesMemorySize);
    hipMemcpy(deviceXforces, sim->xForces.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceYforces, particlesMemorySize);
    hipMemcpy(deviceYforces, sim->yForces.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceZforces, particlesMemorySize);
    hipMemcpy(deviceZforces, sim->zForces.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&deviceMasses, particlesMemorySize);
    hipMemcpy(deviceMasses, sim->masses.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
}

void updateData() {
    hipMemcpy(sim->xForces.data(), deviceXforces,
        particlesMemorySize, hipMemcpyDeviceToHost);
    hipMemcpy(sim->yForces.data(), deviceYforces,
        particlesMemorySize, hipMemcpyDeviceToHost);
    hipMemcpy(sim->zForces.data(), deviceZforces,
        particlesMemorySize, hipMemcpyDeviceToHost);
    
    hipMemcpy(deviceXpos, sim->xPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    hipMemcpy(deviceYpos, sim->yPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
    hipMemcpy(deviceZpos, sim->zPositions.data(),
        particlesMemorySize, hipMemcpyHostToDevice);
}

void cleanupCuda() {
     hipFree(deviceXpos);
     hipFree(deviceYpos);
     hipFree(deviceZpos);
     hipFree(deviceXforces);
     hipFree(deviceYforces);
     hipFree(deviceZforces);
     hipFree(deviceMasses);
}
